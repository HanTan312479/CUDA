/*-----------------------------------------------
 * 请在此处填写你的个人信息
 * 学号:SA24218177
 * 姓名:niujiahao
 * 邮箱:hantan312479@gmail.com
 ------------------------------------------------*/

#include <chrono>
#include <cstring>
#include <fstream>
#include <iostream>
#include <string>
#include <hip/hip_runtime.h>

// 宏定义用于简化访问3D数组
#define AT_GPU(x, y, z) universe[(x) * N * N + (y) * N + z]
#define NEXT_GPU(x, y, z) next[(x) * N * N + (y) * N + z]
#define AT(x, y, z) universe[(x) * N * N + (y) * N + z]

using namespace std;

// 存活细胞数
int population(int N, char *universe)
{
    int result = 0;
    for (int i = 0; i < N * N * N; i++)
        result += universe[i];
    return result;
}

// 打印世界状态
void print_universe(int N, char *universe)
{
    // 仅在N较小(<= 32)时用于Debug
    if (N > 32)
        return;
    for (int x = 0; x < N; x++)
    {
        for (int y = 0; y < N; y++)
        {
            for (int z = 0; z < N; z++)
            {
                if (AT(x, y, z))
                    cout << "O ";
                else
                    cout << "* ";
            }
            cout << endl;
        }
        cout << endl;
    }
    cout << "population: " << population(N, universe) << endl;
}

// CUDA 核函数
__global__ void life3d_kernel(int N, char *universe, char *next) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    // 仅处理在有效范围内的线程
    if (x >= N || y >= N || z >= N) return;

    // 计算当前细胞的存活邻居数量
    int alive = 0;
    for (int dx = -1; dx <= 1; dx++) {
        for (int dy = -1; dy <= 1; dy++) {
            for (int dz = -1; dz <= 1; dz++) {
                if (dx == 0 && dy == 0 && dz == 0) continue;

                // 循环边界处理
                int nx = (x + dx + N) % N;
                int ny = (y + dy + N) % N;
                int nz = (z + dz + N) % N;

                alive += AT_GPU(nx, ny, nz);
            }
        }
    }

    // 状态转移规则
    if (AT_GPU(x, y, z) && (alive < 5 || alive > 7))
        NEXT_GPU(x, y, z) = 0;
    else if (!AT_GPU(x, y, z) && alive == 6)
        NEXT_GPU(x, y, z) = 1;
    else
        NEXT_GPU(x, y, z) = AT_GPU(x, y, z);
}

void life3d_run(int N, char *universe, int T) {
    char *d_universe, *d_next;
    size_t size = N * N * N * sizeof(char);

    // 分配设备内存
    hipMalloc((void **)&d_universe, size);
    hipMalloc((void **)&d_next, size);

    // 将数据从主机传输到设备
    hipMemcpy(d_universe, universe, size, hipMemcpyHostToDevice);

    // 设置 CUDA 网格和块的大小
    dim3 blockSize(8, 8, 8);  // 每个线程块 8x8x8 = 512 个线程
    dim3 gridSize((N + blockSize.x - 1) / blockSize.x,
                  (N + blockSize.y - 1) / blockSize.y,
                  (N + blockSize.z - 1) / blockSize.z);

    for (int t = 0; t < T; t++) {
        // 调用核函数
        life3d_kernel<<<gridSize, blockSize>>>(N, d_universe, d_next);

        // 交换 universe 和 next 的指针
        char *temp = d_universe;
        d_universe = d_next;
        d_next = temp;
    }

    // 将数据从设备传回主机
    hipMemcpy(universe, d_universe, size, hipMemcpyDeviceToHost);

    // 释放设备内存
    hipFree(d_universe);
    hipFree(d_next);
}
// 核心计算代码，将世界向前推进T个时刻
// void life3d_run(int N, char *universe, int T)
// {
//     char *next = (char *)malloc(N * N * N);
//     for (int t = 0; t < T; t++)
//     {
//         // outerloop: iter universe
//         for (int x = 0; x < N; x++)
//             for (int y = 0; y < N; y++)
//                 for (int z = 0; z < N; z++)
//                 {
//                     // inner loop: stencil
//                     int alive = 0;
//                     for (int dx = -1; dx <= 1; dx++)
//                         for (int dy = -1; dy <= 1; dy++)
//                             for (int dz = -1; dz <= 1; dz++)
//                             {
//                                 if (dx == 0 && dy == 0 && dz == 0)
//                                     continue;
//                                 int nx = (x + dx + N) % N;
//                                 int ny = (y + dy + N) % N;
//                                 int nz = (z + dz + N) % N;
//                                 alive += AT(nx, ny, nz);
//                             }
//                     if (AT(x, y, z) && (alive < 5 || alive > 7))
//                         next[x * N * N + y * N + z] = 0;
//                     else if (!AT(x, y, z) && alive == 6)
//                         next[x * N * N + y * N + z] = 1;
//                     else
//                         next[x * N * N + y * N + z] = AT(x, y, z);
//                 }
//         memcpy(universe, next, N * N * N);
//     }
//     free(next);
// }

// 读取输入文件
void read_file(char *input_file, char *buffer)
{
    ifstream file(input_file, std::ios::binary | std::ios::ate);
    if (!file.is_open())
    {
        cout << "Error: Could not open file " << input_file << std::endl;
        exit(1);
    }
    std::streamsize file_size = file.tellg();
    file.seekg(0, std::ios::beg);
    if (!file.read(buffer, file_size))
    {
        std::cerr << "Error: Could not read file " << input_file << std::endl;
        exit(1);
    }
    file.close();
}

// 写入输出文件
void write_file(char *output_file, char *buffer, int N)
{
    ofstream file(output_file, std::ios::binary | std::ios::trunc);
    if (!file)
    {
        cout << "Error: Could not open file " << output_file << std::endl;
        exit(1);
    }
    file.write(buffer, N * N * N);
    file.close();
}

int main(int argc, char **argv)
{
    // cmd args
    if (argc < 5)
    {
        cout << "usage: ./life3d N T input output" << endl;
        return 1;
    }
    int N = std::stoi(argv[1]);
    int T = std::stoi(argv[2]);
    char *input_file = argv[3];
    char *output_file = argv[4];

    char *universe = (char *)malloc(N * N * N);
    read_file(input_file, universe);

    int start_pop = population(N, universe);
    auto start_time = std::chrono::high_resolution_clock::now();
    life3d_run(N, universe, T);
    auto end_time = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> duration = end_time - start_time;
    int final_pop = population(N, universe);
    write_file(output_file, universe, N);

    cout << "start population: " << start_pop << endl;
    cout << "final population: " << final_pop << endl;
    double time = duration.count();
    cout << "time: " << time << "s" << endl;
    cout << "cell per sec: " << T / time * N * N * N << endl;

    free(universe);
    return 0;
}
